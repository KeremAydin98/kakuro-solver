#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#include <fstream>
#include <sstream>
#include <vector>

#include <bits/stdc++.h>
#include <array>

using namespace std;

enum direction {d_down, d_right, none};

#define COORD pair<int, int>

//#define DEBUG

int iter = 0;

//////////////////////////////////////////////
//Auxiliary functions for preparing problem //
//////////////////////////////////////////////

void display_arr(int* arr, int n){

  cout << "arr: ";

  for(int i = 0; i < n; i++){
    cout << arr[i] << " ";
  }

  cout << endl;
  
}

void print_coords(COORD start, COORD end){

  cout << "Start:" << start.first << "," << start.second << endl;
  cout << "End:" << end.first << "," << end.second << endl;
  
}

int find_length(COORD start, COORD end, direction dir){

  if(dir == d_down)
    return end.first - start.first;
  if(dir == d_right)
    return end.second - start.second;

  return -1;
}

void convert_sol(int** mat, int** &sol_mat, int m, int n){

  sol_mat = new int*[m]; //Rows
  for(int i = 0; i < m; i++){
    sol_mat[i] = new int[n]; //Cols
  }

  for(int i = 0; i < m; i++){
    for(int j = 0; j < m; j++){
      if(mat[i][j] == -2)
	sol_mat[i][j] = -2; //Empty value cell
      else
	sol_mat[i][j] = -1; //Hint or empty cell
    }
  }
}

void print_one_matrix(int** matrix, int m, int n){
  cout << "Matrix: " << endl;
  for(int i = 0; i < m; i++){ //rows
    for(int j = 0; j < n; j++){ //cols
      cout << matrix[i][j] << "\t";
    }
    cout << "\n";
  }
}

///Auxiliary functions

struct sum{
  COORD start;
  COORD end;

  int hint;
  int dir;
  int length;
  int* arr;

  void print_sum(){
    cout << "############################" << endl;
    cout << "Creating sum with: " << endl;
    print_coords(start, end);
    cout << "Hint: " << hint << endl;
    cout << "Direction: " << dir << endl;
    cout << "Length: " << length << endl;
    cout << "############################" << endl;
  }
  
  sum(COORD _start, COORD _end, int _hint, direction _dir):
    start(_start), end(_end), hint(_hint), dir(_dir)
  {
    length = find_length(_start, _end, _dir);
    arr = new int[length];
    #ifdef DEBUG
    cout << "############################" << endl;
    cout << "Creating sum with: " << endl;
    print_coords(start, end);
    cout << "Hint: " << hint << endl;
    cout << "Direction: " << dir << endl;
    cout << "Length: " << length << endl;
    cout << "############################" << endl;
    #endif
  }
  
  //~sum(){
  //delete arr;
  //}
};


COORD find_end(int** matrix, int m, int n, int i, int j, direction dir){ //0 down 1 right

  if(dir == d_right){
    for(int jj = j+1; jj < n; jj++){
      if(matrix[i][jj] != -2 || jj == n - 1){
        if(matrix[i][jj] == -2 && jj == n -1)
          jj++;
        COORD END = COORD(i, jj);
        return END;
      }
    }
  }
  else{
    for(int ii = i+1; ii < m; ii++){
      if(matrix[ii][j] != -2 || ii == m - 1){
        if(matrix[ii][j] == -2 && ii == m - 1)
          ii++;
        COORD END = COORD(ii, j);
        return END;
      }
    }
  }
  
}


vector<sum> get_sums(int** matrix, int m, int n){

  vector<sum> sums;
  
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      int val = matrix[i][j];
      if(val != -1 && val != -2){
	int hint = val;
	hint = hint / 10;

	if((hint%100) == 0){
	  hint = (int)(hint/100);
	  COORD START = COORD(i, j+1); 
	  COORD END = find_end(matrix, m, n, i, j, d_right);
	  sum _sum = sum(START, END, hint, d_right);
	  sums.push_back(_sum);
	}

	else{
	  int div = (int)(hint/100);
	  int rem = (int)(hint%100);
   
	  if(div == 0 && rem != 0){
	    COORD START = COORD(i+1,j);
	    COORD END = find_end(matrix, m, n, i, j, d_down);
	    sum _sum = sum(START, END, rem, d_down);
	    sums.push_back(_sum);
	  }

	  if(div != 0 && rem != 0){
	    COORD START1 = COORD(i+1,j);
	    COORD START2 = COORD(i,j+1);
	    COORD END1 = find_end(matrix, m, n, i, j, d_down);
	    COORD END2 = find_end(matrix, m, n, i, j, d_right);
	    sum _sum1 = sum(START1, END1, rem, d_down);
	    sum _sum2 = sum(START2, END2, div, d_right);
	    sums.push_back(_sum1);
	    sums.push_back(_sum2);
	  }
	}
      }

      
    }
  }
  return sums;
}
  

void read_matrix(int** &matrix, ifstream &afile, int m, int n){

  matrix = new int*[m]; //rows

  for(int i = 0; i < m; i++){
    matrix[i] = new int[n]; //cols
  }

  int val;
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      afile >> val;
      matrix[i][j] = val;
    }
  }
}

void sol_to_file(int** mat, int** sol_mat, int m, int n){

  string fname = "visualize.kakuro";
  ofstream to_write(fname);

  to_write << m << " " << n << "\n";

  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      if(mat[i][j] != -2)
	to_write << mat[i][j] << " ";
      else
	to_write << sol_mat[i][j] << " ";
    }
    to_write << "\n";
  }

  to_write.close();
}

//////////////////////////////////////////////
//Auxiliary functions for preparing problem //
//////////////////////////////////////////////

///////////////////////////////////////////////////
//Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////

void flatten_sums(vector<sum> sums, int* h_sum_starts_x, int* h_sum_starts_y, int* h_sum_ends_x, int* h_sum_ends_y, int* h_sum_hints, int* h_sum_lengths, int* h_sum_dirs, int no_sums){

  for(int i = 0; i < no_sums; i++){
    
    h_sum_starts_x[i] = sums[i].start.first;
    h_sum_starts_y[i] = sums[i].start.second;

    h_sum_ends_x[i] = sums[i].end.first;
    h_sum_ends_y[i] = sums[i].end.second;
    
    h_sum_hints[i] = sums[i].hint;
    h_sum_lengths[i] = sums[i].length;
    
    h_sum_dirs[i] = sums[i].dir;
  }
  
}

void print_flattened(int* h_sum_starts_x, int* h_sum_starts_y, int* h_sum_ends_x, int* h_sum_ends_y, int* h_sum_hints, int* h_sum_lengths, int* h_sum_dirs, int no_sums){

  cout << "###h_sum_starts_x: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_starts_x[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_starts_y: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_starts_y[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_ends_x: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_ends_x[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_ends_y: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_ends_y[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_hints: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_hints[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_lengths: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_lengths[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_dirs: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_dirs[i] << " ";
  }
  cout << endl;
  
}

void flatten_sol_mat(int** sol_mat, int* h_sol_mat, int m, int n){
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      h_sol_mat[i*n+j] = sol_mat[i][j];
    }
  }  
}

void print_flattened_matrix(int* h_sol_mat, int m, int n){

  cout << "###Flattened matrix: " << endl;
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      cout << h_sol_mat[i*n+j] << " ";
    }
    cout << endl;
  }
  cout << endl;
}

///////////////////////////////////////////////////
//Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////


///////////////////
//CUDA FUNCTIONS //
///////////////////

void init_iteration(int* iteration, int* sol_mat, int m, int n)
{
  iteration = new int[m*n];

  for(int i=0; i<m; i++){
    for(int j=0; j<n; j++){
      if(sol_mat[i*n+j] == -2){
        iteration[i*n+j] = 0;
      }
      else{
        iteration[i*n+j] = -1;
      }
    }
  }
}

vector<int> remove_unusable_values(int* d_sol_mat, int i, int j, int k,
                                   int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x,
                                   int* d_sum_ends_y, int* d_sum_hints,
                                   int m, int n, vector<sum> sums){

  vector<int> possible_values = {1,2,3,4,5,6,7,8,9};

  // smaller than minimum hint
  // different than the values in row and column
  // minimum value according to length
  vector<int> will_remove = {};
  for(int kk=d_sum_starts_x[k]; kk<d_sum_ends_x[k]; kk++){
    will_remove.push_back(d_sol_mat[i * n + j]);
  }
  for(int ll=d_sum_starts_x[k]; ll<d_sum_ends_x[k]; ll++){
    will_remove.push_back(d_sol_mat[i * n + j]);
  }
  for(int mm=0; mm<possible_values.size(); mm++)
  {
    if((possible_values[mm] >= d_sum_hints[0] || possible_values[mm] >= d_sum_hints[1])){
      will_remove.push_back(possible_values[mm]);
    }
  }

  vector<int>::iterator it;
  for (it = will_remove.begin(); it != will_remove.end(); it++) {
      possible_values.erase(remove(possible_values.begin(), possible_values.end(), *it), possible_values.end());
  }

  return possible_values;
}

void fill_sum(int* d_sol_mat, vector<int> possible_values, int* d_sum_hints, int* d_sum_dirs, int i, int j, int k, int m, int n){

  int summation = 0;
  
  if(d_sum_dirs[k] == 0){

    for(int kk=0; kk<j;kk++){
      if(d_sol_mat[kk * n + j] != -1){ 
        summation += d_sol_mat[kk * n + j];
      }
    }
  }
  else{

    for(int kk=0; kk<j;kk++){
      if(d_sol_mat[i * n + kk] != -1){ 
        summation += d_sol_mat[i * n + kk];
      }
    }
  }

  int last_value = d_sum_hints[k] - summation;

  if(find(possible_values.begin(), possible_values.end(), last_value) != possible_values.end()){

    d_sol_mat[i * n + j] = last_value;

  }
  
}

bool hasRepetitiveValues(const std::vector<int>& vec) {
    std::set<int> uniqueElements;
    for (int element : vec) {
        if (uniqueElements.count(element) > 0) {
            // Element already exists in the set
            return true;
        }
        uniqueElements.insert(element);
    }
    return false;
}

bool check_solution(int* d_sol_mat, int no_sums, int m, int n,
                    int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x,
                    int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs){
  /*
  Confirms the solution to see if it is correct or not
  */
  for(int k=0; k<no_sums; k++){
    int summation = 0;
    vector<int> repetitive_or_not;
    if(d_sum_dirs[k] == 0)
    {
      for(int j=d_sum_starts_x[k]; j<d_sum_ends_x[k]; j++)
      {
        summation += d_sol_mat[d_sum_starts_y[k] * n + j];
        repetitive_or_not.push_back(d_sol_mat[d_sum_starts_y[k] * n + j]);
      }
    }
    else{
      for(int j=d_sum_starts_y[k]; j<d_sum_ends_y[k]; j++)
      {
        summation += d_sol_mat[j * n + d_sum_starts_x[k]];
        repetitive_or_not.push_back(d_sol_mat[d_sum_starts_y[k] * n + j]);
      }
    }

    if(d_sum_hints[k] != summation){
        return false;
    }

    if (hasRepetitiveValues(repetitive_or_not)){
      return false;
    }
  }

  return true;

}

__global__ void kakuro_kernel(int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x,
                              int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs, 
                              int* d_sol_mat, int m, int n, int no_sums, volatile bool* solved,
                              int* iteration, vector<sum> sums){

  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if(tid < no_sums)
  {
    for(int i=d_sum_starts_x[tid]; i<d_sum_ends_x[tid]; i++)
    {

      for(int j=d_sum_starts_y[tid]; j<d_sum_ends_y[tid]; j++)
      {
 
        vector<int> possible_values = remove_unusable_values(d_sol_mat, i, j, tid,
                                                             d_sum_starts_x, d_sum_starts_y, d_sum_ends_x,
                                                             d_sum_ends_y, d_sum_hints,
                                                             m, n, sums);

        if(possible_values.size() == 0){
          iteration[i * n + j] += 1;
          continue;
        }

        if((i == d_sum_ends_x[tid]) && (d_sum_dirs[tid] == 0)){
          int first_value = d_sol_mat[i * n + j];
          fill_sum(d_sol_mat, possible_values, d_sum_hints, i, j, tid, m, n);
          if(d_sol_mat[i * n + j] != first_value){
            continue;
          }
        }
        else if((i == d_sum_ends_y[tid]) && (d_sum_dirs[tid] == 1)){
          int first_value = d_sol_mat[i * n + j];
          fill_sum(d_sol_mat, possible_values, d_sum_hints, i, j, tid, m, n);
          
          if(d_sol_mat[i * n + j] != first_value){
            continue;
          }
        }

        int which_value = iteration[i * n + j] % possible_values.size();


        d_sol_mat[i * n + j] = possible_values[which_value];

        iteration[i * n + j] += 1;

        if(check_solution(d_sol_mat, no_sums, m, n,
                          d_sum_starts_x, d_sum_starts_y, d_sum_ends_x,
                          d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs)){
            *solved = true;
        }
      }
    }
  } 
}
  //About volatile bool* solved:
  //You can get idea from https://stackoverflow.com/questions/12505750/how-can-a-global-function-return-a-value-or-break-out-like-c-c-does%5B/url%5D for how to break out of a CUDA kernel
  //You may or may not use it


///////////////////
//CUDA FUNCTIONS //
///////////////////

int main(int argc, char** argv){
  
  string filename(argv[1]);
  ifstream file;
  file.open(filename.c_str());

  int m, n;
  file >> m;
  file >> n;

  int** mat;
  read_matrix(mat, file, m, n);
  print_one_matrix(mat, m, n);
  
  int** sol_mat;
  convert_sol(mat, sol_mat, m, n);
  print_one_matrix(sol_mat, m, n);
  
  vector<sum> sums = get_sums(mat, m, n);
  
  //CUDA
  hipDeviceProp_t prop; // hipDeviceProp_t prop; declares a variable prop of type hipDeviceProp_t, which is a structure that holds information about a CUDA device.
  hipGetDeviceProperties(&prop, 0); // retrieves the properties of the CUDA device with the device ID 0 and stores the information in the prop variable
  printf("==prop== Running on device: %d -- %s \n", 0, prop.name);
  printf("==prop== #of SM -- %d \n", prop.multiProcessorCount);
  printf("==prop== Max Threads Per Block: -- %d \n", prop.maxThreadsPerBlock);

  //To DO 
  // =========================================
  int BLOCK_SIZE = 16; 
  int GRID_SIZE = (int)ceil(n/BLOCK_SIZE);;  

  // Use dim3 objects
  dim3 grid_dim(GRID_SIZE, GRID_SIZE);
  dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
  // =========================================

  int no_sums = sums.size();

  //Flattening sums and matrix
  int* h_sum_starts_x = new int[no_sums];
  int* h_sum_starts_y = new int[no_sums];
  int* h_sum_ends_x = new int[no_sums];
  int* h_sum_ends_y = new int[no_sums];
  int* h_sum_hints = new int[no_sums];
  int* h_sum_lengths = new int[no_sums];
  int* h_sum_dirs = new int[no_sums];

  // Pair to integers
  flatten_sums(sums, h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

  // Print flattened vector
  print_flattened(h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

  int* h_sol_mat;
  h_sol_mat = new int[m*n];
  flatten_sol_mat(sol_mat, h_sol_mat, m, n);

  print_flattened_matrix(h_sol_mat, m, n);

  //Declare device pointers and copy data into device
  int *d_sum_starts_x, *d_sum_starts_y, *d_sum_ends_x, *d_sum_ends_y, *d_sum_hints, *d_sum_lengths, *d_sum_dirs, *d_sol_mat, *d_t_mats;


  hipMalloc(&d_sum_starts_x, no_sums*sizeof(int));
  hipMalloc(&d_sum_starts_y, no_sums*sizeof(int));
  hipMalloc(&d_sum_ends_x, no_sums*sizeof(int));
  hipMalloc(&d_sum_ends_y, no_sums*sizeof(int));
  hipMalloc(&d_sum_hints, no_sums*sizeof(int));
  hipMalloc(&d_sum_lengths, no_sums*sizeof(int));
  hipMalloc(&d_sum_dirs, no_sums*sizeof(int));
  hipMalloc(&d_sol_mat, (m*n)*sizeof(int));
  hipMalloc(&d_t_mats, (m * n * grid_dim * block_dim)*sizeof(int)); //Allocating invidual matrix for each GPU thread
  //You may use this array if you will implement a thread-wise solution

  hipMemcpy(d_sum_starts_x, h_sum_starts_x, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_starts_y, h_sum_starts_y, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_ends_x, h_sum_ends_x, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_ends_y, h_sum_ends_y, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_hints, h_sum_hints, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_lengths, h_sum_lengths, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_dirs, h_sum_dirs, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sol_mat, h_sol_mat, (m*n)*sizeof(int), hipMemcpyHostToDevice);

  
  bool* solved;
  *solved = false;
  bool* d_solved;
  
  hipMalloc(&d_solved, sizeof(bool));
  hipMemcpy(d_solved, solved, sizeof(bool), hipMemcpyHostToDevice);
  

  // ITERATION MATRIX
  int* iteration;
  init_iteration(iteration, d_sol_mat, m, n);
  // ==============================
  // CUDA kernel
  kakuro_kernel<<<grid_dim, block_dim>>>(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints,
	 				 d_sum_lengths, d_sum_dirs, d_sol_mat, d_t_mats, m, n,
					 no_sums, d_solved, iteration, sums);
  // ===============================
  hipDeviceSynchronize();
  //CUDA
  
  
  print_flattened_matrix(d_sol_mat, m, n);
  //TO DO sol_mat_flattened_to_file(mat, d_sol_mat, m, n)
  //Similiar to sol_mat, use hints from mat and values from d_sol_mat
  
  for(int i = 0; i < n; i++){
    delete mat[i];
    delete sol_mat[i];
  }

  delete mat;
  delete sol_mat;
  
  delete h_sum_starts_x;
  delete h_sum_starts_y;
  delete h_sum_ends_x;
  delete h_sum_ends_y;
  delete h_sum_hints;
  delete h_sum_lengths;
  delete h_sum_dirs;
  delete h_sol_mat;

  hipFree(d_t_mats);
  hipFree(d_sum_starts_x);
  hipFree(d_sum_starts_y);
  hipFree(d_sum_ends_x);
  hipFree(d_sum_ends_y);
  hipFree(d_sum_hints);
  hipFree(d_sum_lengths);
  hipFree(d_sum_dirs);
  hipFree(d_sol_mat);
  
  
  return 0;
}
