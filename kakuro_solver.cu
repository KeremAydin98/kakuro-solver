
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

#include <fstream>
#include <sstream>
#include <vector>

#include <bits/stdc++.h>
#include <array>

using namespace std;

enum direction {d_down, d_right, none};

#define COORD pair<int, int>

//#define DEBUG

int iter = 0;

//////////////////////////////////////////////
//Auxiliary functions for preparing problem //
//////////////////////////////////////////////

void display_arr(int* arr, int n){

  cout << "arr: ";

  for(int i = 0; i < n; i++){
    cout << arr[i] << " ";
  }

  cout << endl;
  
}

void print_coords(COORD start, COORD end){

  cout << "Start:" << start.first << "," << start.second << endl;
  cout << "End:" << end.first << "," << end.second << endl;
  
}

int find_length(COORD start, COORD end, direction dir){

  if(dir == d_down)
    return end.first - start.first;
  if(dir == d_right)
    return end.second - start.second;

  return -1;
}

void convert_sol(int** mat, int** &sol_mat, int m, int n){

  sol_mat = new int*[m]; //Rows
  for(int i = 0; i < m; i++){
    sol_mat[i] = new int[n]; //Cols
  }

  for(int i = 0; i < m; i++){
    for(int j = 0; j < m; j++){
      if(mat[i][j] == -2)
	sol_mat[i][j] = -2; //Empty value cell
      else
	sol_mat[i][j] = -1; //Hint or empty cell
    }
  }
}

void print_one_matrix(int** matrix, int m, int n){
  cout << "Matrix: " << endl;
  for(int i = 0; i < m; i++){ //rows
    for(int j = 0; j < n; j++){ //cols
      cout << matrix[i][j] << "\t";
    }
    cout << "\n";
  }
}

///Auxiliary functions

struct sum{
  COORD start;
  COORD end;

  int hint;
  int dir;
  int length;
  int* arr;

  void print_sum(){
    cout << "############################" << endl;
    cout << "Creating sum with: " << endl;
    print_coords(start, end);
    cout << "Hint: " << hint << endl;
    cout << "Direction: " << dir << endl;
    cout << "Length: " << length << endl;
    cout << "############################" << endl;
  }
  
  sum(COORD _start, COORD _end, int _hint, direction _dir):
    start(_start), end(_end), hint(_hint), dir(_dir)
  {
    length = find_length(_start, _end, _dir);
    arr = new int[length];
    #ifdef DEBUG
    cout << "############################" << endl;
    cout << "Creating sum with: " << endl;
    print_coords(start, end);
    cout << "Hint: " << hint << endl;
    cout << "Direction: " << dir << endl;
    cout << "Length: " << length << endl;
    cout << "############################" << endl;
    #endif
  }
  
  //~sum(){
  //delete arr;
  //}
};


COORD find_end(int** matrix, int m, int n, int i, int j, direction dir){ //0 down 1 right

  if(dir == d_right){
    for(int jj = j+1; jj < n; jj++){
      if(matrix[i][jj] != -2 || jj == n - 1){
        if(matrix[i][jj] == -2 && jj == n -1)
            jj++;
          COORD END = COORD(i, jj);
          return END;

      }
    }
  }
  else{// if(dir == d_down)
    for(int ii = i+1; ii < m; ii++){
      if(matrix[ii][j] != -2 || ii == m - 1){
        if(matrix[ii][j] == -2 && ii == m - 1)
                ii++;
              COORD END = COORD(ii, j);
              return END;
        }
      }
  }

  // no matching end found, return a default value
  COORD not_found = COORD(-1, -1);
  return not_found;
  
}


vector<sum> get_sums(int** matrix, int m, int n){

  vector<sum> sums;
  
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      int val = matrix[i][j];
      if(val != -1 && val != -2){
	int hint = val;
	hint = hint / 10;

	if((hint%100) == 0){
	  hint = (int)(hint/100);
	  COORD START = COORD(i, j+1); 
	  COORD END = find_end(matrix, m, n, i, j, d_right);
	  sum _sum = sum(START, END, hint, d_right);
	  sums.push_back(_sum);
	}

	else{
	  int div = (int)(hint/100);
	  int rem = (int)(hint%100);
   
	  if(div == 0 && rem != 0){
	    COORD START = COORD(i+1,j);
	    COORD END = find_end(matrix, m, n, i, j, d_down);
	    sum _sum = sum(START, END, rem, d_down);
	    sums.push_back(_sum);
	  }

	  if(div != 0 && rem != 0){
	    COORD START1 = COORD(i+1,j);
	    COORD START2 = COORD(i,j+1);
	    COORD END1 = find_end(matrix, m, n, i, j, d_down);
	    COORD END2 = find_end(matrix, m, n, i, j, d_right);
	    sum _sum1 = sum(START1, END1, rem, d_down);
	    sum _sum2 = sum(START2, END2, div, d_right);
	    sums.push_back(_sum1);
	    sums.push_back(_sum2);
	  }
	}
      }

      
    }
  }
  return sums;
}
  

void read_matrix(int** &matrix, ifstream &afile, int m, int n){

  matrix = new int*[m]; //rows

  for(int i = 0; i < m; i++){
    matrix[i] = new int[n]; //cols
  }

  int val;
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      afile >> val;
      matrix[i][j] = val;
    }
  }
}

void sol_to_file(int** mat, int** sol_mat, int m, int n){

  string fname = "visualize.kakuro";
  ofstream to_write(fname);

  to_write << m << " " << n << "\n";

  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      if(mat[i][j] != -2)
	to_write << mat[i][j] << " ";
      else
	to_write << sol_mat[i][j] << " ";
    }
    to_write << "\n";
  }

  to_write.close();
}

//////////////////////////////////////////////
//Auxiliary functions for preparing problem //
//////////////////////////////////////////////

///////////////////////////////////////////////////
//Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////

void flatten_sums(vector<sum> sums, int* h_sum_starts_x, int* h_sum_starts_y, int* h_sum_ends_x, int* h_sum_ends_y, int* h_sum_hints, int* h_sum_lengths, int* h_sum_dirs, int no_sums){

  for(int i = 0; i < no_sums; i++){
    
    h_sum_starts_x[i] = sums[i].start.first;
    h_sum_starts_y[i] = sums[i].start.second;

    h_sum_ends_x[i] = sums[i].end.first;
    h_sum_ends_y[i] = sums[i].end.second;
    
    h_sum_hints[i] = sums[i].hint;
    h_sum_lengths[i] = sums[i].length;
    
    h_sum_dirs[i] = sums[i].dir;
  }
  
}

void print_flattened(int* h_sum_starts_x, int* h_sum_starts_y, int* h_sum_ends_x, int* h_sum_ends_y, int* h_sum_hints, int* h_sum_lengths, int* h_sum_dirs, int no_sums){

  cout << "###h_sum_starts_x: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_starts_x[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_starts_y: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_starts_y[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_ends_x: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_ends_x[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_ends_y: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_ends_y[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_hints: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_hints[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_lengths: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_lengths[i] << " ";
  }
  cout << endl;

  cout << "###h_sum_dirs: " << endl;
  for(int i = 0; i < no_sums; i++){
    cout << h_sum_dirs[i] << " ";
  }
  cout << endl;
  
}

void flatten_sol_mat(int** sol_mat, int* d_sol_mat, int m, int n){
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      d_sol_mat[i*n+j] = sol_mat[i][j];
    }
  }  
}

void print_flattened_matrix(int* d_sol_mat, int m, int n){

  cout << "###Flattened matrix: " << endl;
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      cout << d_sol_mat[i*n+j] << " ";
    }
    cout << endl;
  }
  cout << endl;
}

///////////////////////////////////////////////////
//Auxiliary functions for preparing CUDA setting //
///////////////////////////////////////////////////


///////////////////
//CUDA FUNCTIONS //
///////////////////

void init_iteration(int** iteration, int** sol_mat, int m, int n){

  for(int i = 0; i < m; i++){
    iteration[i] = new int[n];
  }
  for(int i=0; i<m; i++){
    for(int j=0; j<n; j++){
      if(sol_mat[i][j] == -2){
        iteration[i][j] = 0;
      }
      else{
        iteration[i][j] = -1;
      }
    }
  }
}

void flatten_iteration(int** h_iteration, int* d_iteration, int m, int n){
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      d_iteration[i*n+j] = h_iteration[i][j];
    }
  }  
}

/*

__device__ vector<int> remove_unusable_values(int* d_sol_mat, int i, int j, int k,
                                   int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x,
                                   int* d_sum_ends_y, int* d_sum_hints,
                                   int m, int n, vector<sum> sums){

  vector<int> possible_values = {1,2,3,4,5,6,7,8,9};

  // smaller than minimum hint
  // different than the values in row and column
  // minimum value according to length
  std::vector<int> will_remove = {};
  for(int kk=d_sum_starts_x[k]; kk<d_sum_ends_x[k]; kk++){
    will_remove.push_back(d_sol_mat[i * n + j]);
  }
  for(int ll=d_sum_starts_x[k]; ll<d_sum_ends_x[k]; ll++){
    will_remove.push_back(d_sol_mat[i * n + j]);
  }
  for(int mm=0; mm<possible_values.size(); mm++)
  {
    if((possible_values[mm] >= d_sum_hints[0] || possible_values[mm] >= d_sum_hints[1])){
      will_remove.push_back(possible_values[mm]);
    }
  }

  vector<int>::iterator it;
  for (it = will_remove.begin(); it != will_remove.end(); it++) {
      possible_values.erase(remove(possible_values.begin(), possible_values.end(), *it), possible_values.end());
  }

  return possible_values;
}

__device__ void fill_sum(int* d_sol_mat, vector<int> possible_values, int* d_sum_hints, int* d_sum_dirs, int i, int j, int k, int m, int n){

  int summation = 0;
  
  if(d_sum_dirs[k] == 0){

    for(int kk=0; kk<j;kk++){
      if(d_sol_mat[kk * n + j] != -1){ 
        summation += d_sol_mat[kk * n + j];
      }
    }
  }
  else{

    for(int kk=0; kk<j;kk++){
      if(d_sol_mat[i * n + kk] != -1){ 
        summation += d_sol_mat[i * n + kk];
      }
    }
  }

  int last_value = d_sum_hints[k] - summation;

  if(find(possible_values.begin(), possible_values.end(), last_value) != possible_values.end()){

    d_sol_mat[i * n + j] = last_value;

  }
  
}

__device__ bool hasRepetitiveValues(const std::vector<int>& vec) {
    std::set<int> uniqueElements;
    for (int element : vec) {
        if (uniqueElements.count(element) > 0) {
            // Element already exists in the set
            return true;
        }
        uniqueElements.insert(element);
    }
    return false;
}

__device__ bool check_solution(int* d_sol_mat, int no_sums, int m, int n,
                    int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x,
                    int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths, int* d_sum_dirs){
  
  Confirms the solution to see if it is correct or not
  
  for(int k=0; k<no_sums; k++){
    int summation = 0;
    vector<int> repetitive_or_not;
    if(d_sum_dirs[k] == 0)
    {
      for(int j=d_sum_starts_x[k]; j<d_sum_ends_x[k]; j++)
      {
        summation += d_sol_mat[d_sum_starts_y[k] * n + j];
        repetitive_or_not.push_back(d_sol_mat[d_sum_starts_y[k] * n + j]);
      }
    }
    else{
      for(int j=d_sum_starts_y[k]; j<d_sum_ends_y[k]; j++)
      {
        summation += d_sol_mat[j * n + d_sum_starts_x[k]];
        repetitive_or_not.push_back(d_sol_mat[d_sum_starts_y[k] * n + j]);
      }
    }

    if(d_sum_hints[k] != summation){
        return false;
    }

    if (hasRepetitiveValues(repetitive_or_not)){
      return false;
    }
  }

  return true;

}


__global__ void kakuro_kernel(int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x,
                              int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths,
                              int* d_sum_dirs, int* d_sol_mat, 
                              int m, int n, int no_sums, volatile bool* solved,
                              int* d_iteration, vector<sum> sums){

  
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if(tid < no_sums)
  {
    for(int i=d_sum_starts_x[tid]; i<d_sum_ends_x[tid]; i++)
    {

      for(int j=d_sum_starts_y[tid]; j<d_sum_ends_y[tid]; j++)
      {
 
        vector<int> possible_values = remove_unusable_values(d_sol_mat, i, j, tid,
                                                             d_sum_starts_x, d_sum_starts_y, 
                                                             d_sum_ends_x,
                                                             d_sum_ends_y, d_sum_hints,
                                                             m, n, sums);

        if(possible_values.size() == 0){
          d_iteration[i * n + j] += 1;
          continue;
        }

        if((i == d_sum_ends_x[tid]) && (d_sum_dirs[tid] == 0)){
          int first_value = d_sol_mat[i * n + j];
          fill_sum(d_sol_mat, possible_values, d_sum_hints, d_sum_dirs, i, j, tid, m, n);
          if(d_sol_mat[i * n + j] != first_value){
            continue;
          }
        }
        else if((i == d_sum_ends_y[tid]) && (d_sum_dirs[tid] == 1)){
          int first_value = d_sol_mat[i * n + j];
          fill_sum(d_sol_mat, possible_values, d_sum_hints, d_sum_dirs, i, j, tid, m, n);
          
          if(d_sol_mat[i * n + j] != first_value){
            continue;
          }
        }

        int which_value = d_iteration[i * n + j] % possible_values.size();


        d_sol_mat[i * n + j] = possible_values[which_value];

        d_iteration[i * n + j] += 1;

        if(check_solution(d_sol_mat, no_sums, m, n,
                          d_sum_starts_x, d_sum_starts_y, d_sum_ends_x,
                          d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs)){
            *solved = true;
        }
      }
    }
  } 
}
  //About volatile bool* solved:
  //You can get idea from https://stackoverflow.com/questions/12505750/how-can-a-global-function-return-a-value-or-break-out-like-c-c-does%5B/url%5D for how to break out of a CUDA kernel
  //You may or may not use it


///////////////////
//CUDA FUNCTIONS //
///////////////////


__global__ void kakuro_kernel(int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x,
                              int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths,
                              int* d_sum_dirs, int* d_sol_mat, 
                              int m, int n, int no_sums, volatile bool* solved,
                              int* d_iteration, vector<sum> sums){

  
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if(tid < no_sums)
  {
    for(int i=d_sum_starts_x[tid]; i<d_sum_ends_x[tid]; i++)
    {

      for(int j=d_sum_starts_y[tid]; j<d_sum_ends_y[tid]; j++)
      {
 
        // ==============================================================
        vector<int> possible_values = {1,2,3,4,5,6,7,8,9};

        // smaller than minimum hint
        // different than the values in row and column
        // minimum value according to length
        std::vector<int> will_remove = {};
        for(int kk=d_sum_starts_x[tid]; kk<d_sum_ends_x[tid]; kk++){
          will_remove.push_back(d_sol_mat[i * n + j]);
        }
        for(int ll=d_sum_starts_x[tid]; ll<d_sum_ends_x[tid]; ll++){
          will_remove.push_back(d_sol_mat[i * n + j]);
        }
        for(int mm=0; mm<possible_values.size(); mm++)
        {
          if((possible_values[mm] >= d_sum_hints[0] || possible_values[mm] >= d_sum_hints[1])){
            will_remove.push_back(possible_values[mm]);
          }
        }

        for (int value : will_remove) {
            possible_values.erase(remove(possible_values.begin(), possible_values.end(), value), possible_values.end());
        }

        // =============================================================

        if(possible_values.size() == 0){
          d_iteration[i * n + j] += 1;
          continue;
        }

        if((i == d_sum_ends_x[tid]) && (d_sum_dirs[tid] == 0)){
          int first_value = d_sol_mat[i * n + j];

          // =============================================================
          int summation = 0;
          
          if(d_sum_dirs[tid] == 0){

            for(int kk=0; kk<j;kk++){
              if(d_sol_mat[kk * n + j] != -1){ 
                summation += d_sol_mat[kk * n + j];
              }
            }
          }
          else{

            for(int kk=0; kk<j;kk++){
              if(d_sol_mat[i * n + kk] != -1){ 
                summation += d_sol_mat[i * n + kk];
              }
            }
          }

          int last_value = d_sum_hints[tid] - summation;

          if(find(possible_values.begin(), possible_values.end(), last_value) != possible_values.end()){

            d_sol_mat[i * n + j] = last_value;

          }
          
          // =============================================================
          if(d_sol_mat[i * n + j] != first_value){
            continue;
          }
        }
        else if((i == d_sum_ends_y[tid]) && (d_sum_dirs[tid] == 1)){
          int first_value = d_sol_mat[i * n + j];
          // =============================================================
          int summation = 0;
          
          if(d_sum_dirs[tid] == 0){

            for(int kk=0; kk<j;kk++){
              if(d_sol_mat[kk * n + j] != -1){ 
                summation += d_sol_mat[kk * n + j];
              }
            }
          }
          else{

            for(int kk=0; kk<j;kk++){
              if(d_sol_mat[i * n + kk] != -1){ 
                summation += d_sol_mat[i * n + kk];
              }
            }
          }

          int last_value = d_sum_hints[tid] - summation;

          if(find(possible_values.begin(), possible_values.end(), last_value) != possible_values.end()){

            d_sol_mat[i * n + j] = last_value;

          }
          // =============================================================
          if(d_sol_mat[i * n + j] != first_value){
            continue;
          }
        }

        int which_value = d_iteration[i * n + j] % possible_values.size();


        d_sol_mat[i * n + j] = possible_values[which_value];

        d_iteration[i * n + j] += 1;

        // =============================================================
        bool condition = true;
        for(int k=0; k<no_sums; k++){
          int summation = 0;
          vector<int> repetitive_or_not;
          if(d_sum_dirs[k] == 0)
          {
            for(int j=d_sum_starts_x[k]; j<d_sum_ends_x[k]; j++)
            {
              summation += d_sol_mat[d_sum_starts_y[k] * n + j];
              repetitive_or_not.push_back(d_sol_mat[d_sum_starts_y[k] * n + j]);
            }
          }
          else{
            for(int j=d_sum_starts_y[k]; j<d_sum_ends_y[k]; j++)
            {
              summation += d_sol_mat[j * n + d_sum_starts_x[k]];
              repetitive_or_not.push_back(d_sol_mat[d_sum_starts_x[k] * n + j]);
            }
          }

          if(d_sum_hints[k] != summation){
              condition = false;
          }

          bool has_repetitive = false;
          set<int> uniqueElements;
          for (int element : repetitive_or_not) {
              if (uniqueElements.count(element) > 0) {
                  // Element already exists in the set
                  has_repetitive = true;
              }
              uniqueElements.insert(element);
          }

          if (has_repetitive){
            condition = false;
          }
        }

        if(condition){
            *solved = true;
        }
        // =============================================================
      }
    }
  } 
}
*/
__global__ void kakuro_kernel(int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x,
                              int* d_sum_ends_y, int* d_sum_hints, int* d_sum_lengths,
                              int* d_sum_dirs, int* d_sol_mat, 
                              int m, int n, int no_sums, volatile bool* solved,
                              int* d_iteration) {
  
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < no_sums) {
    for (int i = d_sum_starts_x[tid]; i < d_sum_ends_x[tid]; i++) {

      for (int j = d_sum_starts_y[tid]; j < d_sum_ends_y[tid]; j++) {
 
        // ==============================================================
        int possible_values[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
        int possible_values_size = sizeof(possible_values) / sizeof(int);

        // smaller than minimum hint
        // different than the values in row and column
        // minimum value according to length
        int will_remove[9];
        int will_remove_count = 0;
        for (int kk = d_sum_starts_x[tid]; kk < d_sum_ends_x[tid]; kk++) {
          will_remove[will_remove_count++] = d_sol_mat[i * n + j];
        }
        for (int ll = d_sum_starts_x[tid]; ll < d_sum_ends_x[tid]; ll++) {
          will_remove[will_remove_count++] = d_sol_mat[i * n + j];
        }
        for (int mm = 0; mm < possible_values_size; mm++) {
          if ((possible_values[mm] >= d_sum_hints[0] || possible_values[mm] >= d_sum_hints[1])) {
            will_remove[will_remove_count++] = possible_values[mm];
          }
        }

        for (int k = 0; k < will_remove_count; k++) {
          int value = will_remove[k];
          for (int idx = 0; idx < possible_values_size; idx++) {
            if (possible_values[idx] == value) {
              for (int m = idx + 1; m < possible_values_size; m++) {
                possible_values[m - 1] = possible_values[m];
              }
              possible_values_size--;
              break;
            }
          }
        }
        // =============================================================

        if (possible_values_size == 0) {
          atomicAdd(&d_iteration[i * n + j], 1);
          continue;
        }

        if ((i == d_sum_ends_x[tid]) && (d_sum_dirs[tid] == 0)) {
          int first_value = d_sol_mat[i * n + j];

          // =============================================================
          int summation = 0;
          if (d_sum_dirs[tid] == 0) {
            for (int kk = 0; kk < j; kk++) {
              if (d_sol_mat[kk * n + j] != -1) { 
                summation += d_sol_mat[kk * n + j];
              }
            }
          }
          else {
            for (int kk = 0; kk < j; kk++) {
              if (d_sol_mat[i * n + kk] != -1) { 
                summation += d_sol_mat[i * n + kk];
              }
            }
          }

          int last_value = d_sum_hints[tid] - summation;

          bool found = false;
          for (int idx = 0; idx < possible_values_size; idx++) {
            if (possible_values[idx] == last_value) {
              found = true;
              break;
            }
          }

          if (found) {
            d_sol_mat[i * n + j] = last_value;
          }
          // =============================================================
          if (d_sol_mat[i * n + j] != first_value) {
            continue;
          }
        }
        else if ((i == d_sum_ends_y[tid]) && (d_sum_dirs[tid] == 1)) {
          int first_value = d_sol_mat[i * n + j];
          // =============================================================
          int summation = 0;
          if (d_sum_dirs[tid] == 0) {
            for (int kk = 0; kk < j; kk++) {
              if (d_sol_mat[kk * n + j] != -1) { 
                summation += d_sol_mat[kk * n + j];
              }
            }
          }
          else {
            for (int kk = 0; kk < j; kk++) {
              if (d_sol_mat[i * n + kk] != -1) { 
                summation += d_sol_mat[i * n + kk];
              }
            }
          }

          int last_value = d_sum_hints[tid] - summation;

          bool found = false;
          for (int idx = 0; idx < possible_values_size; idx++) {
            if (possible_values[idx] == last_value) {
              found = true;
              break;
            }
          }

          if (found) {
            d_sol_mat[i * n + j] = last_value;
          }
          // =============================================================
          if (d_sol_mat[i * n + j] != first_value) {
            continue;
          }
        }

        int which_value = d_iteration[i * n + j] % possible_values_size;

        d_sol_mat[i * n + j] = possible_values[which_value];

        atomicAdd(&d_iteration[i * n + j], 1);

        // =============================================================
        bool condition = true;
        for (int k = 0; k < no_sums; k++) {
          int summation = 0;
          int repetitive_or_not[100];  // Assuming a maximum size of repetitive_or_not
          int repetitive_or_not_count = 0;
          if (d_sum_dirs[k] == 0) {
            for (int j = d_sum_starts_x[k]; j < d_sum_ends_x[k]; j++) {
              summation += d_sol_mat[d_sum_starts_y[k] * n + j];
              repetitive_or_not[repetitive_or_not_count++] = d_sol_mat[d_sum_starts_y[k] * n + j];
            }
          }
          else {
            for (int j = d_sum_starts_y[k]; j < d_sum_ends_y[k]; j++) {
              summation += d_sol_mat[j * n + d_sum_starts_x[k]];
              repetitive_or_not[repetitive_or_not_count++] = d_sol_mat[d_sum_starts_x[k] * n + j];
            }
          }

          if (d_sum_hints[k] != summation) {
            condition = false;
          }

          bool has_repetitive = false;
          for (int i = 0; i < repetitive_or_not_count; i++) {
            for (int j = i + 1; j < repetitive_or_not_count; j++) {
              if (repetitive_or_not[i] == repetitive_or_not[j]) {
                has_repetitive = true;
                break;
              }
            }
            if (has_repetitive) {
              break;
            }
          }

          if (has_repetitive) {
            condition = false;
          }
        }

        if (condition) {
          *solved = true;
        }
        // =============================================================
      }
    }
  } 
}


int main(int argc, char** argv){
  
  string filename(argv[1]);
  ifstream file;
  file.open(filename.c_str());

  int m, n;
  file >> m;
  file >> n;

  int** mat;
  read_matrix(mat, file, m, n);
  print_one_matrix(mat, m, n);
  
  int** h_sol_mat;
  convert_sol(mat, h_sol_mat, m, n);
  print_one_matrix(h_sol_mat, m, n);
  
  vector<sum> sums = get_sums(mat, m, n);
  
  //CUDA
  hipDeviceProp_t prop; // cudaDeviceProp prop; declares a variable prop of type cudaDeviceProp, which is a structure that holds information about a CUDA device.
  hipGetDeviceProperties(&prop, 0); // retrieves the properties of the CUDA device with the device ID 0 and stores the information in the prop variable
  printf("==prop== Running on device: %d -- %s \n", 0, prop.name);
  printf("==prop== #of SM -- %d \n", prop.multiProcessorCount);
  printf("==prop== Max Threads Per Block: -- %d \n", prop.maxThreadsPerBlock);

  //To DO 
  // =========================================
  int block_dim = 1024; 
  int grid_dim = (int)ceil(n/block_dim);;  

  // =========================================

  int no_sums = sums.size();

  //Flattening sums and matrix
  int* h_sum_starts_x = new int[no_sums];
  int* h_sum_starts_y = new int[no_sums];
  int* h_sum_ends_x = new int[no_sums];
  int* h_sum_ends_y = new int[no_sums];
  int* h_sum_hints = new int[no_sums];
  int* h_sum_lengths = new int[no_sums];
  int* h_sum_dirs = new int[no_sums];

  // Pair to integers
  flatten_sums(sums, h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

  // Print flattened vector
  print_flattened(h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

  int* d_sol_mat;
  int* d_iteration = new int[m*n];
  d_sol_mat = new int[m*n];
  flatten_sol_mat(h_sol_mat, d_sol_mat, m, n);

  print_flattened_matrix(d_sol_mat, m, n);

  //Declare device pointers and copy data into device
  int *d_sum_starts_x, *d_sum_starts_y, *d_sum_ends_x, *d_sum_ends_y, *d_sum_hints, *d_sum_lengths, *d_sum_dirs, *d_t_mats;

  // ITERATION MATRIX
  int** h_iteration = new int*[m];
  init_iteration(h_iteration, h_sol_mat, m, n);
  flatten_iteration(h_iteration, d_iteration, m, n);
  // ==============================

  hipMalloc(&d_sum_starts_x, no_sums*sizeof(int));
  hipMalloc(&d_sum_starts_y, no_sums*sizeof(int));
  hipMalloc(&d_sum_ends_x, no_sums*sizeof(int));
  hipMalloc(&d_sum_ends_y, no_sums*sizeof(int));
  hipMalloc(&d_sum_hints, no_sums*sizeof(int));
  hipMalloc(&d_sum_lengths, no_sums*sizeof(int));
  hipMalloc(&d_sum_dirs, no_sums*sizeof(int));
  hipMalloc(&d_sol_mat, (m*n)*sizeof(int));
  hipMalloc(&d_iteration, (m*n)*sizeof(int));
  hipMalloc(&d_t_mats, (m * n * grid_dim * block_dim)*sizeof(int)); //Allocating invidual matrix for each GPU thread
  //You may use this array if you will implement a thread-wise solution

  hipMemcpy(d_sum_starts_x, h_sum_starts_x, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_starts_y, h_sum_starts_y, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_ends_x, h_sum_ends_x, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_ends_y, h_sum_ends_y, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_hints, h_sum_hints, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_lengths, h_sum_lengths, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sum_dirs, h_sum_dirs, no_sums*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sol_mat, h_sol_mat, (m*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_iteration, h_iteration, (m*n)*sizeof(int), hipMemcpyHostToDevice);
  
  bool* solved = new bool;
  *solved = false;
  bool* d_solved = new bool;
  
  hipMalloc(&d_solved, sizeof(bool));
  hipMemcpy(d_solved, solved, sizeof(bool), hipMemcpyHostToDevice);
  

  
  
  // CUDA kernel
  kakuro_kernel<<<grid_dim, block_dim>>>(d_sum_starts_x, d_sum_starts_y, d_sum_ends_x,
                                         d_sum_ends_y, d_sum_hints, d_sum_lengths, d_sum_dirs, 
                                         d_sol_mat, m, n, no_sums, d_solved,
                                         d_iteration);
  // ===============================
  hipDeviceSynchronize();
  //CUDA
  
  
  print_flattened_matrix(d_sol_mat, m, n);
  //TO DO sol_mat_flattened_to_file(mat, d_sol_mat, m, n)
  //Similiar to sol_mat, use hints from mat and values from d_sol_mat
  
  for(int i = 0; i < n; i++){
    delete mat[i];
    delete h_sol_mat[i];
  }

  delete mat;
  delete h_sol_mat;
  delete d_iteration;

  delete h_sum_starts_x;
  delete h_sum_starts_y;
  delete h_sum_ends_x;
  delete h_sum_ends_y;
  delete h_sum_hints;
  delete h_sum_lengths;
  delete h_sum_dirs;
  delete d_sol_mat;

  hipFree(d_t_mats);
  hipFree(d_sum_starts_x);
  hipFree(d_sum_starts_y);
  hipFree(d_sum_ends_x);
  hipFree(d_sum_ends_y);
  hipFree(d_sum_hints);
  hipFree(d_sum_lengths);
  hipFree(d_sum_dirs);
  hipFree(d_sol_mat);
  hipFree(d_iteration);
  
  
  return 0;
}
